#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
              
#define N 1000
#define I 100000

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// GPU - CUDA kernel A
__global__ void device_kernel_A(int *A)
{
    int sum = 0;
    for(int n = 0; n < N; n++){
        sum += A[N*n + threadIdx.x];
    }
}

// GPU - CUDA kernel B
__global__ void device_kernel_B(int *B)
{
    int sum = 0;
    for(int n = 0; n < N; n++){
        sum += B[N*threadIdx.x + n];
    }
}

double walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

int main(int argc, char **argv)
{
    // Walltime variables
    double timeStart;
    double timeA = 0;
    double timeB = 0;

    // Host variables
    int a[N*N];
    int b[N*N];
    bool testA = true;
    bool testB = true;

    // Set some random values
    for(int y = 0; y < N; y++){
        for(int x = 0; x < N; x++){
            a[y*N + x] = x + y;
            b[y*N + x] = x + y;
        }
    }

    //********************************* GPU work start *********************************
    
    // Variables
    int *A = NULL;
    int *B = NULL;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&A, N*N * sizeof(int)));
    cudaErrorCheck(hipMalloc((void**)&B, N*N * sizeof(int)));
    
    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(A, a, N*N * sizeof(int), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(B, b, N*N * sizeof(int), hipMemcpyHostToDevice));

    if(testA){
        // GPU computation A
        for(int i = 0; i < I; i++){
            timeStart = walltime();
            device_kernel_A<<<1,N>>>(A);
            timeA += walltime() - timeStart;
        }
    }

    if(testB){
        // GPU computation B
        for(int i = 0; i < I; i++){
            timeStart = walltime();
            device_kernel_B<<<1,N>>>(B);
            timeB += walltime() - timeStart;
        }
    }

    // Free the device memory
    cudaErrorCheck(hipFree(A));
    cudaErrorCheck(hipFree(B));

    //********************************* GPU work stop *********************************

    printf("\n");
    printf("Walltime A: %7.3f ms \n", timeA * 1e3);
    printf("Walltime B: %7.3f ms \n", timeB * 1e3);
    printf("\n");

    return 0;
};
