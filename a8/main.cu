#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
              
#define N       512      
#define I       100000
#define BLOCKS  1
#define ORDER   1

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// GPU - CUDA kernel A
__global__ void device_kernel_A(int *A)
{
    int sum = 0;
    for(int i = 0; i < N; i++){
        sum += A[i*N + threadIdx.x];
    }
}

// GPU - CUDA kernel B
__global__ void device_kernel_B(int *B)
{
    int sum = 0;
    for(int i = 0; i < N; i++){
        sum += B[N*threadIdx.x + i];
    }
}

double walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

int main(int argc, char **argv)
{
    // Walltime variables
    double timeStart;
    double timeA;
    double timeB;

    // Host variables
    int a[N*N];
    int b[N*N];
    bool testA = true;
    bool testB = true;

    // Set some random values
    for(int y = 0; y < N; y++){
        for(int x = 0; x < N; x++){
            a[y*N + x] = x + y;
            b[y*N + x] = x + y;
        }
    }

    //********************************* GPU work start *********************************
    
    // Variables
    int *A = NULL;
    int *B = NULL;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&A, N*N * sizeof(int)));
    cudaErrorCheck(hipMalloc((void**)&B, N*N * sizeof(int)));
    
    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(A, a, N*N * sizeof(int), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(B, b, N*N * sizeof(int), hipMemcpyHostToDevice));
    
    // Warm up        
    device_kernel_A<<<BLOCKS,N>>>(A);
    device_kernel_B<<<BLOCKS,N>>>(B);

    if(ORDER == 1){
        if(testA){
            // GPU computation A
            timeStart = walltime();
            for(int i = 0; i < I; i++){
                device_kernel_A<<<BLOCKS,N>>>(A);
            }
            timeA = walltime() - timeStart;
        }
        if(testB){
            // GPU computation B
            timeStart = walltime();
            for(int i = 0; i < I; i++){
                device_kernel_B<<<BLOCKS,N>>>(B);
            }
            timeB = walltime() - timeStart;
        }
    }
    else{
        if(testB){
            // GPU computation B
            timeStart = walltime();
            for(int i = 0; i < I; i++){
                device_kernel_B<<<BLOCKS,N>>>(B);
            }
            timeB = walltime() - timeStart;
        }
        if(testA){
            // GPU computation A
            timeStart = walltime();
            for(int i = 0; i < I; i++){
                device_kernel_A<<<BLOCKS,N>>>(A);
            }
            timeA = walltime() - timeStart;
        }
    }

    // Free the device memory
    cudaErrorCheck(hipFree(A));
    cudaErrorCheck(hipFree(B));

    //********************************* GPU work stop *********************************

    printf("\n");
    printf("Walltime A: %7.3f ms \n", timeA * 1e3);
    printf("Walltime B: %7.3f ms \n", timeB * 1e3);
    printf("\n");

    return 0;
};
