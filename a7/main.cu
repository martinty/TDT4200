#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
extern "C" { 
    #include "libs/bitmap.h"
}

namespace cg = cooperative_groups;

// BLOCKX * BLOCKY can max be 1024 (threads)
// GRIDX * GRIDY can max be 28 (MSs)
#define BLOCKY  32
#define BLOCKX  32
#define GRIDX   7
#define GRIDY   4 

#define ERROR_EXIT -1
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// Constant memory for GPU
__constant__ int constFilterGPU[25];

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

/*
int const sobelYFilter[] = {-1, -2, -1,
                            0, 0, 0,
                            1, 2, 1};
float const sobelYFilterFactor = (float)1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2, 0, -2,
                            -1, 0, -1, 0};
float const sobelXFilterFactor = (float)1.0;
*/

int const laplacian1Filter[] = {-1, -4, -1,
                                -4, 20, -4,
                                -1, -4, -1};

float const laplacian1FilterFactor = (float)1.0;

/*
int const laplacian2Filter[] = {0, 1, 0,
                                1, -4, 1,
                                0, 1, 0};
float const laplacian2FilterFactor = (float)1.0;

int const laplacian3Filter[] = {-1, -1, -1,
                                -1, 8, -1,
                                -1, -1, -1};
float const laplacian3FilterFactor = (float)1.0;

// Bonus Filter:
int const gaussianFilter[] = {1, 4, 6, 4, 1,
                              4, 16, 24, 16, 4,
                              6, 24, 36, 24, 6,
                              4, 16, 24, 16, 4,
                              1, 4, 6, 4, 1};
float const gaussianFilterFactor = (float)1.0 / 256.0;
*/

// CPU serial - Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, const int width, const int height, 
                 const int *filter, const int filterDim, const float filterFactor)
{
    const int filterCenter = filterDim / 2;
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int aggregate = 0;
            for (int ky = 0; ky < filterDim; ky++)
            {
                int nky = filterDim - 1 - ky;
                for (int kx = 0; kx < filterDim; kx++)
                {
                    int nkx = filterDim - 1 - kx;

                    int yy = y + (ky - filterCenter);
                    int xx = x + (kx - filterCenter);
                    if (xx >= 0 && xx < width && yy >= 0 && yy < height)
                        aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
                }
            }
            aggregate *= filterFactor;
            if (aggregate > 0)
                out[y][x] = (aggregate > 255) ? 255 : aggregate;
            else
                out[y][x] = 0;
        }
    }
}

// GPU basic - Apply convolutional filter on image data
__global__ void device_applyFilter(unsigned char *out, const unsigned char *in, const int width, const int height, 
                                   const int *filter, const int filterDim, const float filterFactor)
{
    const int x = blockIdx.x * BLOCKX + threadIdx.x;
    const int y = blockIdx.y * BLOCKY + threadIdx.y;
    if (x < width && y < height)
    {
        const int filterCenter = filterDim / 2;
        int aggregate = 0;
        for (int ky = 0; ky < filterDim; ky++)
        {
            int nky = filterDim - 1 - ky;
            for (int kx = 0; kx < filterDim; kx++)
            {
                int nkx = filterDim - 1 - kx;
                int yy = y + (ky - filterCenter);
                int xx = x + (kx - filterCenter);
                if (xx >= 0 && xx < width && yy >= 0 && yy < height)
                    aggregate += in[xx + yy*width] * filter[nky * filterDim + nkx];
            }
        }
        aggregate *= filterFactor;
        if (aggregate > 0)
            out[x + y*width] = (aggregate > 255) ? 255 : aggregate;
        else
            out[x + y*width] = 0;
    }
}

// GPU shared memory - Apply convolutional filter on image data
__global__ void device_applyFilter_sm(unsigned char *out, const unsigned char *in, const int width, const int height, const int N, 
                                      const int *filter, const int filterDim, const float filterFactor)
{
    extern __shared__ unsigned char data_sm[];
    int *filter_sm = (int*)&data_sm[N];
    const int x = blockIdx.x * BLOCKX + threadIdx.x;
    const int y = blockIdx.y * BLOCKY + threadIdx.y;
    const int xLocal = threadIdx.x;
    const int yLocal = threadIdx.y;
    const int filterCenter = (filterDim / 2);

    if(xLocal == 1 && yLocal == 1)
    {
        for (int i = 0; i < filterDim*filterDim; i++)
            filter_sm[i] = filter[i];
    }

    if (x < width && y < height)
    {
        // Tranfer center to shared memory
        data_sm[(xLocal+filterCenter) + (yLocal+filterCenter)*(BLOCKX+filterCenter*2)] = in[x + y*width];
        
        // Transfer west border to shared memory
        if(threadIdx.x == 0)
        {
            for(int i = 1; i <= filterCenter; i++)
            {
                if(threadIdx.y == 0) // North West corner
                {
                    for(int j = 0; j <= filterCenter; j++)
                    {
                        if(x - i >= 0 && y - j >= 0)
                            data_sm[(xLocal+filterCenter-i) + (yLocal+filterCenter-j)*(BLOCKX+filterCenter*2)] = in[x-i + (y-j)*width];
                    }
                }
                else
                {
                    if(x - i >= 0)
                        data_sm[(xLocal+filterCenter-i) + (yLocal+filterCenter)*(BLOCKX+filterCenter*2)] = in[x-i + y*width];
                }
            }
        }
        
        // Transfer north border to shared memory
        if(threadIdx.y == 0)
        {
            for(int i = 1; i <= filterCenter; i++)
            {
                if(threadIdx.x == BLOCKX-1) // North East corner
                {
                    for(int j = 0; j <= filterCenter; j++)
                    {
                        if(x + j < width && y - i >= 0)
                            data_sm[(xLocal+filterCenter+j) + (yLocal+filterCenter-i)*(BLOCKX+filterCenter*2)] = in[x+j + (y-i)*width];
                    }
                }
                else
                {
                    if(y - i >= 0)
                        data_sm[(xLocal+filterCenter) + (yLocal+filterCenter-i)*(BLOCKX+filterCenter*2)] = in[x + (y-i)*width];
                }
            }
        }
        
        // Transfer east border to shared memory
        if(threadIdx.x == BLOCKX-1)
        {
            for(int i = 1; i <= filterCenter; i++)
            {
                if(threadIdx.y == BLOCKY-1) // South East corner
                {
                    for(int j = 0; j <= filterCenter; j++)
                    {
                        if(x + i < width && y + j < height)
                            data_sm[(xLocal+filterCenter+i) + (yLocal+filterCenter+j)*(BLOCKX+filterCenter*2)] = in[x+i + (y+j)*width];
                    }
                }
                else
                {
                    if(x + i < width)
                        data_sm[(xLocal+filterCenter+i) + (yLocal+filterCenter)*(BLOCKX+filterCenter*2)] = in[x+i + y*width];
                }
            }
        }
        
        // Transfer south border to shared memory
        if(threadIdx.y == BLOCKY-1)
        {
            for(int i = 1; i <= filterCenter; i++)
            {
                if(threadIdx.x == 0) // South West corner
                {
                    for(int j = 0; j <= filterCenter; j++)
                    {
                        if(x - j >= 0 && y + i < height)
                            data_sm[(xLocal+filterCenter-j) + (yLocal+filterCenter+i)*(BLOCKX+filterCenter*2)] = in[x-j + (y+i)*width];
                    }
                }
                else
                {
                    if(y + i < height)
                        data_sm[(xLocal+filterCenter) + (yLocal+filterCenter+i)*(BLOCKX+filterCenter*2)] = in[x + (y+i)*width];
                }
            }
        }

        __syncthreads();
        int aggregate = 0;
        for (int ky = 0; ky < filterDim; ky++)
        {
            int nky = filterDim - 1 - ky;
            for (int kx = 0; kx < filterDim; kx++)
            {
                int nkx = filterDim - 1 - kx;
                int yy = yLocal + ky;
                int xx = xLocal + kx;
                int yyy = y + (ky - filterCenter);
                int xxx = x + (kx - filterCenter);
                if (xxx >= 0 && xxx < width && yyy >= 0 && yyy < height)
                    aggregate += data_sm[xx + yy*(BLOCKX+filterCenter*2)] * filter_sm[nky * filterDim + nkx];
            }
        }
        aggregate *= filterFactor;
        if (aggregate > 0)
            out[x + y*width] = (aggregate > 255) ? 255 : aggregate;
        else
            out[x + y*width] = 0;
    }
    else
        __syncthreads();
}

// GPU cooperative groups - Apply convolutional filter on image data
__global__ void device_applyFilter_cg(unsigned char *image, unsigned char *process, const int width, const int height, const int Nx, const int Ny, 
    const int filterDim, const float filterFactor, const int iterations)
{
    //__shared__ unsigned char data_sq[1]; 
    cg::grid_group grid = cg::this_grid();
    for (int i = 0; i < iterations; i++)
    {
        int x = (blockIdx.x * BLOCKX + threadIdx.x) * Nx;
        int y = (blockIdx.y * BLOCKY + threadIdx.y) * Ny;
        for(int iy = 0; iy < Ny; iy++)
        {
            for(int ix = 0; ix < Nx; ix++)
            {
                if (x < width && y < height)
                {
                    const int filterCenter = filterDim / 2;
                    int aggregate = 0;
                    for (int ky = 0; ky < filterDim; ky++)
                    {
                        int nky = filterDim - 1 - ky;
                        for (int kx = 0; kx < filterDim; kx++)
                        {
                        int nkx = filterDim - 1 - kx;
                        int yy = y + (ky - filterCenter);
                        int xx = x + (kx - filterCenter);
                        if (xx >= 0 && xx < width && yy >= 0 && yy < height)
                            aggregate += image[xx + yy*width] * constFilterGPU[nky * filterDim + nkx];
                        }
                    }
                    aggregate *= filterFactor;
                    if (aggregate > 0)
                        process[x + y*width] = (aggregate > 255) ? 255 : aggregate;
                    else
                        process[x + y*width] = 0;
                }
                x++;
            }
            y++;
        }
        cg::sync(grid);
        unsigned char *temp = process;
        *process = *image;
        *image = *temp;
        cg::sync(grid);
    }
}

void help(char const *exec, char const opt, char const *optarg)
{
    FILE *out = stdout;
    if (opt != 0)
    {
        out = stderr;
        if (optarg)
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        else
            fprintf(out, "Invalid parameter - %c\n", opt);
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");
    fprintf(out, "  -t, --test                       compare GPU and CPU code\n");
    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

double walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

int isImageChannelEqual(unsigned char **a, unsigned char **b, const int sizeX, const int sizeY, const int ch)
{
    int errors = 0;
    for(int y = 0; y < sizeY; y++)
    {
        for(int x = 0; x < sizeX; x++)
        {
            if(a[y][x] != b[y][x])
            {
                if(errors == 0)
                    printf("\n*** Pixel errors in image ***\n");
                errors++;
                if(errors <= 5)
                printf("Ch %d: Pixel [x=%d, y=%d] \t expected %d \t got %d\n", ch, x, y, b[y][x], a[y][x]);
            }
        }
    }
    if(errors > 0)
        printf("Ch %d: %d errors!\n", ch, errors);
    return errors;
}

int getNumberOfSM(int devID)
{
    hipDeviceProp_t deviceProp;
    cudaErrorCheck(hipGetDeviceProperties(&deviceProp, devID));
    return deviceProp.multiProcessorCount;
}

void freeMemory(char *output, char *input, bmpImage *image, 
    bmpImageChannel *imageChannel1, bmpImageChannel *imageChannel2, bmpImageChannel *imageChannel3, bmpImageChannel *imageChannel4)
{
    if (output)
        free(output);
    if (input)
        free(input);
    if (image)
        freeBmpImage(image);
    if (imageChannel1)
        freeBmpImageChannel(imageChannel1);
    if (imageChannel2)
        freeBmpImageChannel(imageChannel2);
    if (imageChannel3)
        freeBmpImageChannel(imageChannel3);
    if (imageChannel4)
        freeBmpImageChannel(imageChannel4);    
}

int main(int argc, char **argv)
{
    // Walltime variables
    double startTime;
    double serialTime = 0;
    double cudaTime = 0;
    double cudaTime_sm = 0;
    double cudaTime_cg = 0;
    double activateCudaTime = 0;

    // Compare GPU and CPU code
    bool test = false;

    // Parameter parsing
    unsigned int iterations = 1;
    char *output = NULL;
    char *input = NULL;
    bmpImage *image = NULL;
    bmpImageChannel *imageChannel1 = NULL;
    bmpImageChannel *imageChannel2 = NULL;
    bmpImageChannel *imageChannel3 = NULL;
    bmpImageChannel *imageChannel4 = NULL;

    static struct option const long_options[] = {
        {"help", no_argument, 0, 'h'},
        {"test", no_argument, 0, 't'},
        {"iterations", required_argument, 0, 'i'},
        {0, 0, 0, 0}
    };

    static char const *short_options = "hti:";
    {
        char *endptr;
        int c;
        int option_index = 0;
        while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1)
        {
            switch (c)
            {
            case 'h':
                help(argv[0], 0, NULL);
                return 0;
            case 't':
                test = true;
                break;
            case 'i':
                iterations = strtol(optarg, &endptr, 10);
                if (endptr == optarg)
                {
                    help(argv[0], c, optarg);
                    return ERROR_EXIT;
                }
                break;
            default:
                abort();
            }
        }
    }

    if (argc <= (optind + 1))
    {
        help(argv[0], ' ', "Not enough arugments");
        return ERROR_EXIT;
    }
    input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
    strncpy(input, argv[optind], strlen(argv[optind]));
    optind++;

    output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
    strncpy(output, argv[optind], strlen(argv[optind]));
    optind++;
    // End of parameter parsing!

    // Create the BMP image and load it from disk.
    image = newBmpImage(0, 0);
    if (image == NULL)
    {
        fprintf(stderr, "Could not allocate new image!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }
    if (loadBmpImage(image, input) != 0)
    {
        fprintf(stderr, "Could not load bmp image '%s'!\n", input);
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }

    // sizeX and sizeY for image
    const int sizeX = image->width;
    const int sizeY = image->height;

    // Offset for BLOCK/GRID size sent to the cuda kernel
    int offsetBlockX = 0;
    int offsetBlockY = 0;
    int offsetGridX = 0;
    int offsetGridY = 0;
    if (sizeX % BLOCKX)
        offsetBlockX = 1;
    if (sizeY % BLOCKY)
        offsetBlockY = 1;
    if (sizeX % (GRIDX * BLOCKX))
        offsetGridX = 1;
    if (sizeY % (GRIDY * BLOCKY))
        offsetGridY = 1;
        
    if (test)
    {
        // Create a single color channel image for CPU serial code
        imageChannel1 = newBmpImageChannel(sizeX, sizeY);
        if (imageChannel1 == NULL)
        {
            fprintf(stderr, "Could not allocate new image channel 1!\n");
            freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
            return ERROR_EXIT;
        }
        if (extractImageChannel(imageChannel1, image, extractAverage) != 0)
        {
            fprintf(stderr, "Could not extract image channel 1!\n");
            freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
            return ERROR_EXIT;
        }
    }

    // Create a single color channel image for GPU basic code
    imageChannel2 = newBmpImageChannel(sizeX, sizeY);
    if (imageChannel2 == NULL)
    {
        fprintf(stderr, "Could not allocate new image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }
    if (extractImageChannel(imageChannel2, image, extractAverage) != 0)
    {
        fprintf(stderr, "Could not extract image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }

    // Create a single color channel image for GPU shared memory code
    imageChannel3 = newBmpImageChannel(sizeX, sizeY);
    if (imageChannel3 == NULL)
    {
        fprintf(stderr, "Could not allocate new image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }
    if (extractImageChannel(imageChannel3, image, extractAverage) != 0)
    {
        fprintf(stderr, "Could not extract image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }

    // Create a single color channel image for GPU cooperative groups code
    imageChannel4 = newBmpImageChannel(sizeX, sizeY);
    if (imageChannel3 == NULL)
    {
        fprintf(stderr, "Could not allocate new image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }
    if (extractImageChannel(imageChannel4, image, extractAverage) != 0)
    {
        fprintf(stderr, "Could not extract image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }

    // Activate CUDA - No delay inside work later
    startTime = walltime();
    unsigned char *dummy;
    cudaErrorCheck(hipMalloc((void**)&dummy, sizeof(unsigned char)));
    cudaErrorCheck(hipFree(dummy));
    activateCudaTime = walltime() - startTime;

    // Choose filter
    const int *filter = laplacian1Filter;
    const int filterDim = 3;
    const float filterFactor = laplacian1FilterFactor;

    if (test)
    {
        //********************************* CPU serial work start *********************************
        startTime = walltime();

        // CPU computation!
        bmpImageChannel *processImageChannel = newBmpImageChannel(sizeX, sizeY);
        for (unsigned int i = 0; i < iterations; i++)
        {
            applyFilter(
                processImageChannel->data,
                imageChannel1->data,
                sizeX, sizeY,
                filter, filterDim, filterFactor
            );
            unsigned char **tmp = processImageChannel->data;
            processImageChannel->data = imageChannel1->data;
            imageChannel1->data = tmp;
            unsigned char *tmp_raw = processImageChannel->rawdata;
            processImageChannel->rawdata = imageChannel1->rawdata;
            imageChannel1->rawdata = tmp_raw;
        }
        freeBmpImageChannel(processImageChannel);

        serialTime = walltime() - startTime;
        //********************************* CPU serial work stop *********************************
    }

    //********************************* GPU basic work start *********************************
    startTime = walltime();

    // Variables
    dim3 gridBlock(sizeX/BLOCKX + offsetBlockX, sizeY/BLOCKY + offsetBlockY);
    dim3 threadBlock(BLOCKX, BLOCKY);
    unsigned char *imageChannelGPU = NULL;
    unsigned char *processImageChannelGPU = NULL;
    int *filterGPU = NULL;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&imageChannelGPU, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&processImageChannelGPU, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&filterGPU, filterDim*filterDim * sizeof(int)));

    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(imageChannelGPU, imageChannel2->rawdata, sizeX*sizeY * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(filterGPU, filter, filterDim*filterDim * sizeof(int), hipMemcpyHostToDevice));

    // GPU computation
    for (unsigned int i = 0; i < iterations; i++)
    {
        device_applyFilter<<<gridBlock, threadBlock>>>(
            processImageChannelGPU, 
            imageChannelGPU,
            sizeX, sizeY, 
            filterGPU, filterDim, filterFactor
        );
        cudaErrorCheck(hipGetLastError());
        unsigned char *temp = processImageChannelGPU;
        processImageChannelGPU = imageChannelGPU;
        imageChannelGPU = temp;
    }

    // Copy data from device to host
    cudaErrorCheck(hipMemcpy(imageChannel2->rawdata, imageChannelGPU, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Free the device memory
    cudaErrorCheck(hipFree(imageChannelGPU));
    cudaErrorCheck(hipFree(processImageChannelGPU));
    cudaErrorCheck(hipFree(filterGPU));

    cudaTime = walltime() - startTime;
    //********************************* GPU basic work stop *********************************

    //********************************* GPU shared memory work start *********************************
    startTime = walltime();

    // Variables
    dim3 gridBlock_sm(sizeX/BLOCKX + offsetBlockX, sizeY/BLOCKY + offsetBlockY);
    dim3 threadBlock_sm(BLOCKX, BLOCKY);
    unsigned char *imageChannelGPU_sm = NULL;
    unsigned char *processImageChannelGPU_sm = NULL;
    int *filterGPU_sm = NULL;
    const int N = (BLOCKX+2*(filterDim/2))*(BLOCKY+2*(filterDim/2));
    const int sizeImage_sm = N * sizeof(unsigned char);
    const int sizeFilter_sm = filterDim * filterDim * sizeof(int);
    const int size_sm = sizeImage_sm + sizeFilter_sm;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&imageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&processImageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&filterGPU_sm, filterDim*filterDim * sizeof(int)));

    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(imageChannelGPU_sm, imageChannel3->rawdata, sizeX*sizeY * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(filterGPU_sm, filter, filterDim*filterDim * sizeof(int), hipMemcpyHostToDevice));

    // GPU computation
    for (unsigned int i = 0; i < iterations; i++)
    {
        device_applyFilter_sm<<<gridBlock_sm, threadBlock_sm, size_sm>>>(
            processImageChannelGPU_sm, 
            imageChannelGPU_sm,
            sizeX, sizeY, N, 
            filterGPU_sm, filterDim, filterFactor
        );
        cudaErrorCheck(hipGetLastError());
        unsigned char *temp_sm = processImageChannelGPU_sm;
        processImageChannelGPU_sm = imageChannelGPU_sm;
        imageChannelGPU_sm = temp_sm;
    }

    // Copy data from device to host
    cudaErrorCheck(hipMemcpy(imageChannel3->rawdata, imageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Free the device memory
    cudaErrorCheck(hipFree(imageChannelGPU_sm));
    cudaErrorCheck(hipFree(processImageChannelGPU_sm));
    cudaErrorCheck(hipFree(filterGPU_sm));
    
    cudaTime_sm = walltime() - startTime;
    //********************************* GPU shared memory work stop *********************************

    //********************************* GPU cooperative groups work start ***************************
    startTime = walltime();

    // Variables
    dim3 gridDim_cg(GRIDX, GRIDY);
    dim3 blockDim_cg(BLOCKX, BLOCKY);
    unsigned char *imageChannelGPU_cg = NULL;
    unsigned char *processImageChannelGPU_cg = NULL;
    const int sizeFilter_cg = filterDim * filterDim * sizeof(int);
    const int Nx = sizeX / (GRIDX * BLOCKX) + offsetGridX;
    const int Ny = sizeY / (GRIDY * BLOCKY) + offsetGridY;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&imageChannelGPU_cg, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&processImageChannelGPU_cg, sizeX*sizeY * sizeof(unsigned char)));

    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(imageChannelGPU_cg, imageChannel3->rawdata, sizeX*sizeY * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constFilterGPU), filter, sizeFilter_cg));

    // Arguments for CUDA kernel
    void *kernelArgs_cg[] = {
        (void *)&imageChannelGPU_cg, (void *)&processImageChannelGPU_cg,
        (void *)&sizeX, (void *)&sizeY,
        (void *)&Nx, (void *)&Ny,
        (void *)&filterDim, (void *)&filterFactor,
        (void *)&iterations
    };

    // GPU computation
    cudaErrorCheck(hipLaunchCooperativeKernel(
        (void *)device_applyFilter_cg,
        gridDim_cg, blockDim_cg, 
        kernelArgs_cg,                                      
        0, 
        NULL
    ));
    cudaErrorCheck(hipGetLastError());

    // Copy data from device to host
    cudaErrorCheck(hipMemcpy(imageChannel4->rawdata, imageChannelGPU_cg, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Free the device memory
    cudaErrorCheck(hipFree(imageChannelGPU_cg));
    cudaErrorCheck(hipFree(processImageChannelGPU_cg));
    
    cudaTime_cg = walltime() - startTime;
    //********************************* GPU cooperative groups work stop ****************************

    if (test)
    {
        // Check if GPU image channel is equal to CPU image channel
        int errors = 0;
        errors += isImageChannelEqual(imageChannel2->data, imageChannel1->data, sizeX, sizeY, 2);
        errors += isImageChannelEqual(imageChannel3->data, imageChannel1->data, sizeX, sizeY, 3);
        errors += isImageChannelEqual(imageChannel4->data, imageChannel1->data, sizeX, sizeY, 4);
        if(errors == 0)
            printf("\n*** Every pixel is correct! ***\n");
    }

    // Map our single color image back to a normal BMP image with 3 color channels
    if (mapImageChannel(image, imageChannel4, mapEqual) != 0)
    {
        fprintf(stderr, "Could not map image channel!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    }

    //Write the image back to disk
    if (saveBmpImage(image, output) != 0)
    {
        fprintf(stderr, "Could not save output to '%s'!\n", output);
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
        return ERROR_EXIT;
    };

    printf("\n*** Run times ***\n");
    printf("Running with %d iteration(s)\n", iterations);
    printf("Activate CUDA time:%12.3f ms\n", activateCudaTime * 1e3);
    printf("     Work GPU time:%12.3f ms\tCooperative groups \tch 4 \n", cudaTime_cg * 1e3);
    printf("     Work GPU time:%12.3f ms\tShared memory \t\tch 3 \n", cudaTime_sm * 1e3);
    printf("     Work GPU time:%12.3f ms\tBasic \t\t\tch 2 \n", cudaTime * 1e3);

    if(test)
    {
        printf("     Work CPU time:%12.3f ms\tSerial \t\t\tch 1 \n", serialTime * 1e3);
        printf("\n*** Speedups ***\n");
        printf("Cooperative groups GPU: %6.1f times faster then shared memory GPU\n", cudaTime_sm/cudaTime_cg);
        printf("Cooperative groups GPU: %6.1f times faster then basic GPU\n", cudaTime/cudaTime_cg);
        printf("Cooperative groups GPU: %6.1f times faster then serial CPU\n", serialTime/cudaTime_cg);
        printf("     Shared memory GPU: %6.1f times faster then basic GPU\n", cudaTime/cudaTime_sm);
        printf("     Shared memory GPU: %6.1f times faster then serial CPU\n", serialTime/cudaTime_sm);
        printf("             Basic GPU: %6.1f times faster then serial CPU\n\n", serialTime/cudaTime);
    }
    else
        printf("\n");

    freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3, imageChannel4);
    return 0;
};
