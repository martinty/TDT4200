#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
extern "C" { 
    #include "libs/bitmap.h"
}

// Divide the problem into blocks of BLOCKX x BLOCKY threads
#define BLOCKY 8
#define BLOCKX 8

#define ERROR_EXIT -1
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

/*
int const sobelYFilter[] = {-1, -2, -1,
                            0, 0, 0,
                            1, 2, 1};
float const sobelYFilterFactor = (float)1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2, 0, -2,
                            -1, 0, -1, 0};
float const sobelXFilterFactor = (float)1.0;
*/

int const laplacian1Filter[] = {-1, -4, -1,
                                -4, 20, -4,
                                -1, -4, -1};

float const laplacian1FilterFactor = (float)1.0;

/*
int const laplacian2Filter[] = {0, 1, 0,
                                1, -4, 1,
                                0, 1, 0};
float const laplacian2FilterFactor = (float)1.0;

int const laplacian3Filter[] = {-1, -1, -1,
                                -1, 8, -1,
                                -1, -1, -1};
float const laplacian3FilterFactor = (float)1.0;

// Bonus Filter:
int const gaussianFilter[] = {1, 4, 6, 4, 1,
                              4, 16, 24, 16, 4,
                              6, 24, 36, 24, 6,
                              4, 16, 24, 16, 4,
                              1, 4, 6, 4, 1};
float const gaussianFilterFactor = (float)1.0 / 256.0;
*/

// CPU serial - Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{
    unsigned int const filterCenter = (filterDim / 2);
    for (unsigned int y = 0; y < height; y++)
    {
        for (unsigned int x = 0; x < width; x++)
        {
            int aggregate = 0;
            for (unsigned int ky = 0; ky < filterDim; ky++)
            {
                int nky = filterDim - 1 - ky;
                for (unsigned int kx = 0; kx < filterDim; kx++)
                {
                    int nkx = filterDim - 1 - kx;

                    int yy = y + (ky - filterCenter);
                    int xx = x + (kx - filterCenter);
                    if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
                        aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
                }
            }
            aggregate *= filterFactor;
            if (aggregate > 0)
                out[y][x] = (aggregate > 255) ? 255 : aggregate;
            else
                out[y][x] = 0;
        }
    }
}

// GPU basic - Apply convolutional filter on image data
__global__ void device_applyFilter(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{
    unsigned int x = blockIdx.x * BLOCKX + threadIdx.x;
    unsigned int y = blockIdx.y * BLOCKY + threadIdx.y;
    if (x < width && y < height)
    {
        unsigned int const filterCenter = (filterDim / 2);
        int aggregate = 0;
        for (unsigned int ky = 0; ky < filterDim; ky++)
        {
            int nky = filterDim - 1 - ky;
            for (unsigned int kx = 0; kx < filterDim; kx++)
            {
                int nkx = filterDim - 1 - kx;

                int yy = y + (ky - filterCenter);
                int xx = x + (kx - filterCenter);
                if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
                    aggregate += in[xx + yy*width] * filter[nky * filterDim + nkx];
            }
        }
        aggregate *= filterFactor;
        if (aggregate > 0)
            out[x + y*width] = (aggregate > 255) ? 255 : aggregate;
        else
            out[x + y*width] = 0;
    }
}

// GPU shared memory - Apply convolutional filter on image data
__global__ void device_applyFilter_sm(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{
    //__shared__ unsigned char data[9];
    unsigned int x = blockIdx.x * BLOCKX + threadIdx.x;
    unsigned int y = blockIdx.y * BLOCKY + threadIdx.y;
    if (x < width && y < height)
    {
        //data[x + y*width] = in[x + y*width];
        __syncthreads();
        unsigned int const filterCenter = (filterDim / 2);
        int aggregate = 0;
        for (unsigned int ky = 0; ky < filterDim; ky++)
        {
            int nky = filterDim - 1 - ky;
            for (unsigned int kx = 0; kx < filterDim; kx++)
            {
                int nkx = filterDim - 1 - kx;

                int yy = y + (ky - filterCenter);
                int xx = x + (kx - filterCenter);
                if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
                    aggregate += in[xx + yy*width] * filter[nky * filterDim + nkx];
            }
        }
        aggregate *= filterFactor;
        if (aggregate > 0)
            out[x + y*width] = (aggregate > 255) ? 255 : aggregate;
        else
            out[x + y*width] = 0;
    }
    else
        __syncthreads();
}

void help(char const *exec, char const opt, char const *optarg)
{
    FILE *out = stdout;
    if (opt != 0)
    {
        out = stderr;
        if (optarg)
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        else
            fprintf(out, "Invalid parameter - %c\n", opt);
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");
    fprintf(out, "  -t, --test                       compare GPU and CPU code\n");
    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

double walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}

bool isImageChannelEqual(unsigned char *a, unsigned char *b, unsigned int size)
{
    for (unsigned int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
            return false;
    }
    return true;
}

void freeMemory(char *output, char *input, bmpImage *image, bmpImageChannel *imageChannel1, bmpImageChannel *imageChannel2, bmpImageChannel *imageChannel3)
{
    if (output)
        free(output);
    if (input)
        free(input);
    if (image)
        freeBmpImage(image);
    if (imageChannel1)
        freeBmpImageChannel(imageChannel1);
    if (imageChannel2)
        freeBmpImageChannel(imageChannel2);
    if (imageChannel3)
        freeBmpImageChannel(imageChannel3);
}

int main(int argc, char **argv)
{
    // Walltime variables
    double startTime;
    double serialTime = 0;
    double cudaTime = 0;
    double cudaTime_sm = 0;

    // Compare GPU and CPU code
    bool test = false;

    // Parameter parsing
    unsigned int iterations = 1;
    char *output = NULL;
    char *input = NULL;
    bmpImage *image = NULL;
    bmpImageChannel *imageChannel1 = NULL;
    bmpImageChannel *imageChannel2 = NULL;
    bmpImageChannel *imageChannel3 = NULL;

    static struct option const long_options[] = {
        {"help", no_argument, 0, 'h'},
        {"test", no_argument, 0, 't'},
        {"iterations", required_argument, 0, 'i'},
        {0, 0, 0, 0}
    };

    static char const *short_options = "hti:";
    {
        char *endptr;
        int c;
        int option_index = 0;
        while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1)
        {
            switch (c)
            {
            case 'h':
                help(argv[0], 0, NULL);
                return 0;
            case 't':
                test = true;
                break;
            case 'i':
                iterations = strtol(optarg, &endptr, 10);
                if (endptr == optarg)
                {
                    help(argv[0], c, optarg);
                    return ERROR_EXIT;
                }
                break;
            default:
                abort();
            }
        }
    }

    if (argc <= (optind + 1))
    {
        help(argv[0], ' ', "Not enough arugments");
        return ERROR_EXIT;
    }
    input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
    strncpy(input, argv[optind], strlen(argv[optind]));
    optind++;

    output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
    strncpy(output, argv[optind], strlen(argv[optind]));
    optind++;
    // End of parameter parsing!

    // Create the BMP image and load it from disk.
    image = newBmpImage(0, 0);
    if (image == NULL)
    {
        fprintf(stderr, "Could not allocate new image!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }
    if (loadBmpImage(image, input) != 0)
    {
        fprintf(stderr, "Could not load bmp image '%s'!\n", input);
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }

    // Set sizeX and sizeY for image
    unsigned int sizeX = image->width;
    unsigned int sizeY = image->height;

    if (test)
    {
        // Create a single color channel image for CPU serial code
        imageChannel1 = newBmpImageChannel(sizeX, sizeY);
        if (imageChannel1 == NULL)
        {
            fprintf(stderr, "Could not allocate new image channel 1!\n");
            freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
            return ERROR_EXIT;
        }
        if (extractImageChannel(imageChannel1, image, extractAverage) != 0)
        {
            fprintf(stderr, "Could not extract image channel 1!\n");
            freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
            return ERROR_EXIT;
        }
    }

    // Create a single color channel image for GPU basic code
    imageChannel2 = newBmpImageChannel(sizeX, sizeY);
    if (imageChannel2 == NULL)
    {
        fprintf(stderr, "Could not allocate new image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }
    if (extractImageChannel(imageChannel2, image, extractAverage) != 0)
    {
        fprintf(stderr, "Could not extract image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }

    // Create a single color channel image for GPU shared memory code
    imageChannel3 = newBmpImageChannel(sizeX, sizeY);
    if (imageChannel3 == NULL)
    {
        fprintf(stderr, "Could not allocate new image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }
    if (extractImageChannel(imageChannel3, image, extractAverage) != 0)
    {
        fprintf(stderr, "Could not extract image channel 2!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }

    if (test)
    {
        //********************************* CPU serial work start *********************************
        startTime = walltime();

        // Here we do the actual computation!
        // imageChannel->data is a 2-dimensional array of unsigned char which is accessed row first ([y][x])
        bmpImageChannel *processImageChannel = newBmpImageChannel(sizeX, sizeY);
        for (unsigned int i = 0; i < iterations; i++)
        {
            applyFilter(
                processImageChannel->data,
                imageChannel1->data,
                sizeX,
                sizeY,
                (int *)laplacian1Filter, 3, laplacian1FilterFactor
                //(int *)laplacian2Filter, 3, laplacian2FilterFactor
                //(int *)laplacian3Filter, 3, laplacian3FilterFactor
                //(int *)gaussianFilter, 5, gaussianFilterFactor
            );
            //Swap the data pointers
            unsigned char **tmp = processImageChannel->data;
            processImageChannel->data = imageChannel1->data;
            imageChannel1->data = tmp;
            unsigned char *tmp_raw = processImageChannel->rawdata;
            processImageChannel->rawdata = imageChannel1->rawdata;
            imageChannel1->rawdata = tmp_raw;
        }
        freeBmpImageChannel(processImageChannel);

        serialTime = walltime() - startTime;
        //********************************* CPU serial work stop *********************************
    }

    //********************************* GPU basic work start *********************************
    startTime = walltime();

    // Variables
    dim3 gridBlock(sizeX/BLOCKX, sizeY/BLOCKY);
    dim3 threadBlock(BLOCKX, BLOCKY);
    unsigned char *imageChannelGPU = NULL;
    unsigned char *processImageChannelGPU = NULL;
    int *filterGPU = NULL;
    unsigned int filterDim = 3;
    float filterFactor = laplacian1FilterFactor;
    const int *filter = laplacian1Filter;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&imageChannelGPU, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&processImageChannelGPU, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&filterGPU, filterDim*filterDim * sizeof(int)));

    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(imageChannelGPU, imageChannel2->rawdata, sizeX*sizeY * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(filterGPU, filter, filterDim*filterDim * sizeof(int), hipMemcpyHostToDevice));

    // GPU computation
    for (unsigned int i = 0; i < iterations; i++)
    {
        device_applyFilter<<<gridBlock, threadBlock>>>(
            processImageChannelGPU, 
            imageChannelGPU,
            sizeX,
            sizeY, 
            filterGPU, filterDim, filterFactor
        );
        cudaErrorCheck(hipGetLastError());
        cudaErrorCheck(hipMemcpy(imageChannelGPU, processImageChannelGPU, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToDevice));
    }

    // Copy data from device to host
    cudaErrorCheck(hipMemcpy(imageChannel2->rawdata, imageChannelGPU, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Free the device memory
    cudaErrorCheck(hipFree(imageChannelGPU));
    cudaErrorCheck(hipFree(processImageChannelGPU));
    cudaErrorCheck(hipFree(filterGPU));

    cudaTime = walltime() - startTime;
    //********************************* GPU basic work stop *********************************

    //********************************* GPU shared memory work start *********************************
    startTime = walltime();

    // Variables
    dim3 gridBlock_sm(sizeX/BLOCKX, sizeY/BLOCKY);
    dim3 threadBlock_sm(BLOCKX, BLOCKY);
    unsigned char *imageChannelGPU_sm = NULL;
    unsigned char *processImageChannelGPU_sm = NULL;
    int *filterGPU_sm = NULL;
    unsigned int filterDim_sm = 3;
    float filterFactor_sm = laplacian1FilterFactor;
    const int *filter_sm = laplacian1Filter;

    // Set up device memory
    cudaErrorCheck(hipMalloc((void**)&imageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&processImageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char)));
    cudaErrorCheck(hipMalloc((void**)&filterGPU_sm, filterDim_sm*filterDim_sm * sizeof(int)));

    // Copy data from host to device
    cudaErrorCheck(hipMemcpy(imageChannelGPU_sm, imageChannel3->rawdata, sizeX*sizeY * sizeof(unsigned char), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(filterGPU_sm, filter_sm, filterDim_sm*filterDim_sm * sizeof(int), hipMemcpyHostToDevice));

    // GPU computation
    for (unsigned int i = 0; i < iterations; i++)
    {
        device_applyFilter_sm<<<gridBlock_sm, threadBlock_sm>>>(
            processImageChannelGPU_sm, 
            imageChannelGPU_sm,
            sizeX,
            sizeY, 
            filterGPU_sm, filterDim_sm, filterFactor_sm
        );
        cudaErrorCheck(hipGetLastError());
        cudaErrorCheck(hipMemcpy(imageChannelGPU_sm, processImageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToDevice));
    }

    // Copy data from device to host
    cudaErrorCheck(hipMemcpy(imageChannel3->rawdata, imageChannelGPU_sm, sizeX*sizeY * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Free the device memory
    cudaErrorCheck(hipFree(imageChannelGPU_sm));
    cudaErrorCheck(hipFree(processImageChannelGPU_sm));
    cudaErrorCheck(hipFree(filterGPU_sm));
    
    cudaTime_sm = walltime() - startTime;
    //********************************* GPU shared memory work stop *********************************

    if (test)
    {
        // Check if GPU image channel is equal to CPU image channel
        if (!isImageChannelEqual(imageChannel2->rawdata, imageChannel1->rawdata, sizeX*sizeY) ||
            !isImageChannelEqual(imageChannel3->rawdata, imageChannel1->rawdata, sizeX*sizeY))
        {
            fprintf(stderr, "GPU image channel is not equal to serial image channel!\n");
            freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
            return ERROR_EXIT;
        }
    }

    // Map our single color image back to a normal BMP image with 3 color channels
    // mapEqual puts the color value on all three channels the same way
    // other mapping functions are mapRed, mapGreen, mapBlue
    if (mapImageChannel(image, imageChannel2, mapEqual) != 0)
    {
        fprintf(stderr, "Could not map image channel!\n");
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    }

    //Write the image back to disk
    if (saveBmpImage(image, output) != 0)
    {
        fprintf(stderr, "Could not save output to '%s'!\n", output);
        freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
        return ERROR_EXIT;
    };

    printf("\nGPU time:\t%7.3f s  or  %7.3f ms\tBasic\n", cudaTime, cudaTime * 1e3);
    printf("GPU time:\t%7.3f s  or  %7.3f ms\tShared memory\n", cudaTime_sm, cudaTime_sm * 1e3);
    if (test)
        printf("CPU time:\t%7.3f s  or  %7.3f ms\tSerial\n", serialTime, serialTime * 1e3);

    freeMemory(output, input, image, imageChannel1, imageChannel2, imageChannel3);
    return 0;
};

/*
 // Kernel Function: Runs per device thread
 __global__ void vectorMultUsingSharedMemory(float *dev_a, float, *dev_b, float *dev_c, int width)
{
    const int TILE_WIDTH = 16;
    // Allocate shared memory
    __shared__ float dev_as[TILE_WIDTH][TILE_WIDTH];
    __shared__ float dev_bs[TILE_WIDTH][TILE_WIDTH];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float dotProduct = 0.0f;
    for (int i = 0; i < width/TILE_WIDTH; i++)
    {
        dev_as[threadIdx.y][threadIdx.x] = dev_a[row * width + (i * TILE_WIDTH + threadIdx.x)];
        dev_bs[threadIdx.y][threadIdx.x] = dev_b[(i * TILE_WIDTH + threadIdx.y) * width + col];
        __syncthreads(); 
        for (int j = 0; j < TILE_WIDTH; j++) 
            dotProduct += dev_as[threadIdx.y][j] * dev_bs[j][threadIdx.x];
        __syncthreads();
    }
    dev_c[row * width + col] = dotProduct;
} 
*/